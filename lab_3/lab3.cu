#include <iostream>
#include <fstream>
#include <time.h>
#include <hip/hip_runtime.h>


#include <omp.h>
#include <stdlib.h>
#include <stdio.h>

using namespace std;

/*
40. Найти минимальное число, которое может быть разложено в сумму простых чисел не менее, чем N различными способами (например, для числа 10 существует ровно 5 таких способов: 10 = 7+3 = 5+5 = 5+3+2 = 3+3+2+2 = 2+2+2+2+2)

Как собрать: nvcc -o lab3 -g lab3.cu -Xcompiler "-fopenmp"
*/

__device__ void pullParallel1(int sum, int num, int index, int br, bool stop, bool *mass, int *cnt)
{
	if (!stop)
	{
		sum += num;
		if (sum == index)
		{
			cnt[blockIdx.x * blockDim.x + threadIdx.x] += 1;
			br = true;
			return;
		}
	}
	else
	{
		stop = 0;
		cnt[blockIdx.x * blockDim.x + threadIdx.x] = 0;
		if (sum == index)
			cnt[blockIdx.x * blockDim.x + threadIdx.x] += 1;
	}
	for (int i = num; i + sum <= index; i == 2 ? i++ : i += 2)
		if (mass[i] && i != index)
		{
			pullParallel1(sum, i, index, br, stop, mass, cnt);
			if (br)
				break;
		}
}

void pull(int sum, int num, int index, int br, bool stop, bool *mass, int &cnt)
{
	if (!stop)
	{
		sum += num;
		if (sum == index)
		{
			cnt += 1;
			br = true;
			return;
		}
	}
	else
	{
		stop = 0;
		if (sum == index){
			cnt += 1;
		}
	}
	for (int i = num; i + sum <= index; i == 2 ? i++ : i += 2)
		if (mass[i] && i != index)
		{
			pull(sum, i, index, br, stop, mass, cnt);
			if (br) {
				break;
			}
		}
}

__global__ void start(bool *mass, int i, int *cnt)
{
	pullParallel1(2 * (blockIdx.x * blockDim.x + threadIdx.x) + 2, 3, i, 0, 1, mass, cnt);
}

int sum(int *m, int n)
{
	int s = 0;
	for (int i = 0; i < n; i++)
		s += m[i];
	return s;
}

int searchParallel(bool *mass, int N, int size)
{
	int i = 4, decomp = 0;
	int c = 0;
	bool *mass1;
	hipMalloc((void **)&mass1, size * sizeof(bool));
	hipMemcpy(mass1, mass, size * sizeof(bool), hipMemcpyHostToDevice);
	int *cnt;
	int *count = new int[i / 2];

	while (decomp < N)
	{
		decomp = 0;
		hipMalloc((void **)&cnt, i / 2 * sizeof(int));
		int bl = 1;
		int N1 = i / 2;
		while (N1 > i / 2)
		{
			if (N1 % 2)
				break;
			bl *= 2;
			N1 /= 2;
		}
		start<<<bl, N1>>>(mass1, i, cnt);
		delete[] count;
		count = new int[i / 2];
		hipMemcpy(count, cnt, i / 2 * sizeof(int), hipMemcpyDeviceToHost);
		decomp += sum(count, i / 2);
		hipFree(cnt);
		for (int j = 3; j <= i - 2; j += 2)
			if (mass[j])
				pull(0, j, i, 0, 0, mass, c);
		decomp += c;
		i++;
		c = 0;
	}
	return (i - 1);
}

int search(bool *mass, int N)
{
	int i = 4, decomp = 0, c = 0;
	while (decomp < N)
	{
		decomp = 0;
		for (int k = 2; k <= i; k += 2)
			pull(k, 3, i, 0, 1, mass, c);
		for (int j = 3; j <= i - 2; j += 2)
			if (mass[j])
				pull(0, j, i, 0, 0, mass, c);
		decomp += c;
		c = 0;
		i++;
	}
	return (i - 1);
}

int main()
{
	setlocale(LC_CTYPE, "russian");
	int N, result, result1;
	double time, time1;
	bool *mass1, *mass;
	cout << "Введите число N: ";
	cin >> N;

	int size = log(double(N)) * 10;
	mass1 = new bool[size];
	mass = new bool[size];
	for (int i = 0; i < size; i++)
		mass[i] = mass1[i] = 1;

	for (int i = 2; i < size; i++)
		if (mass1[i])
			for (int j = i * i; j < size; j += i)
				mass[j] = mass1[j] = false;

	cout << "Решение в последовательном варианте." << endl;
	time = omp_get_wtime();
	result = search(mass, N);
	time = omp_get_wtime() - time;
	cout << "Число: " << result << "." << endl;
	cout << "Затраченное время: " << time << "." << endl;
	cout << "Решение в параллельном варианте." << endl;
	time1 = omp_get_wtime();
	result1 = searchParallel(mass1, N, size);
	time1 = omp_get_wtime() - time1;
	cout << "Затраченное время: " << time1 << "." << endl;
	cout << "Число: " << result << "." << endl;
	delete[] mass1;
	return 0;
}
